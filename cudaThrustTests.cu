#include "hip/hip_runtime.h"
/*
If kernels are not running, run deviceQuery and add flag -arch=sm_50 <- or whatever your device cuda capability is
*/

#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "npp.h" // interoperable with NPP types?
#include "thrust/system/cuda/experimental/pinned_allocator.h" // need this specifically

// namespace for experimental allocator is too long
namespace tsce = thrust::system::cuda::experimental;

#define LENGTH 1000000

__global__
void myadd_kernel(float* v, int len)
{
    int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = tIdx; i < len; i = i + stride)
    {
        v[i] = (float)i;
    }
}

int main(int argc, char *argv[])
{
    // Attempt to allocate a simple 32f host vector (pinned + unpinned)
    thrust::host_vector<float, tsce::pinned_allocator<float>> hv_32f(LENGTH);
    thrust::host_vector<float> hv_32f_unpin(LENGTH);

    // And a similar device vector
    thrust::device_vector<float> dv_32f(LENGTH);

    // Raw allocation
    float* darr_32f, *harr_32f;
    hipMalloc((void**)&darr_32f, sizeof(float) * LENGTH);
    hipHostMalloc((void**)&harr_32f, sizeof(float) * LENGTH);

    // Attempt to copy over
    dv_32f = hv_32f_unpin; // note that this still causes async copies? how is that possible?
    dv_32f = hv_32f; // async copy, as viewed in nvvp

    // via explicit 'copy'
    thrust::copy(hv_32f_unpin.begin(), hv_32f_unpin.end(), dv_32f.begin()); // this is still async copy?
    // however, all async copies in the default stream are serialized anyway..

    // Write using a kernel?
    myadd_kernel << <2, 512 >> > ((float*)thrust::raw_pointer_cast(dv_32f.data()), dv_32f.size());
    myadd_kernel << < 2, 512 >> > (darr_32f, LENGTH);

    // Copy it back
    hv_32f = dv_32f;

    for (int i = 0; i < 5; i++)
    {
        printf("%d: %f\n", i, hv_32f[i]);
    }

    // Raw copy back
    hipMemcpy(harr_32f, darr_32f, sizeof(float) * LENGTH, hipMemcpyDeviceToHost);
    for (int i = 0; i < 5; i++)
    {
        printf("%d: %f\n", i, harr_32f[i]);
    }

    // cleanup
    hipFree(darr_32f);
    hipHostFree(harr_32f);

    return 0;
}