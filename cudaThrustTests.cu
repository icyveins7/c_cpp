#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "npp.h" // interoperable with NPP types?
#include "thrust/system/cuda/experimental/pinned_allocator.h" // need this specifically

// namespace for experimental allocator is too long
namespace cuta = thrust::system::cuda::experimental;

#define LENGTH 1000000

int main(int argc, char *argv[])
{
    // Attempt to allocate a simple 32f host vector
    thrust::host_vector<float, cuta::pinned_allocator<float>> hv_32f(LENGTH);

    // And a similar device vector
    thrust::device_vector<float> dv_32f(LENGTH);

    // Attempt to copy over
    dv_32f = hv_32f;

    return 0;
}