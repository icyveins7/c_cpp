#include "hip/hip_complex.h"
// #include "device_functions.h"
#include "hip/hip_runtime.h"


#include <iostream>

/*
This is the simple method of storing complex arrays into shared mem.
It simply packs everything in as-is.
*/
__global__ void cplxAddPackedShared(const hipFloatComplex *x,
                                    const hipFloatComplex *y, hipFloatComplex *z,
                                    int n) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // assign shared memory things
  extern __shared__ double s[];

  // allocate each workspace to be same as blockDim
  // align one after another, i.e. well packed
  hipFloatComplex *s_x = (hipFloatComplex *)s;
  hipFloatComplex *s_y = (hipFloatComplex *)&s_x[blockDim.x];

  s_x[threadIdx.x] = x[idx];
  s_y[threadIdx.x] = y[idx];
  __syncthreads();

  // add and then write out
  z[idx] = hipCaddf(s_x[threadIdx.x], s_y[threadIdx.x]);
}

/*
clang-format off

In this one we add a stagger between the two arrays in shared memory.
Assuming a blockDim of multiples of 32, we would have

  0| 1| 2|...................|30|31
 re|im|re|im.................|re|im
.......................
.......................
NUL|re|im| ................        -> this line is the start of y, which is offset by 1 bank

In hindsight, I think this doesn't matter because during the writes are
separate, and the reads are also separate

clang-format on
*/
__global__ void cplxAddStaggeredShared(const hipFloatComplex *x,
                                       const hipFloatComplex *y,
                                       hipFloatComplex *z, int n) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // assign shared memory things
  extern __shared__ double s[];

  // allocate each workspace to be same as blockDim
  // leave a space of 32-bits after the first one
  float *f_x = (float *)s; // we use float here to count elements
  float *f_gap = (float *)&f_x[2 * blockDim.x];
  float *f_y = (float *)&f_gap[1];

  const hipFloatComplex t_x = x[idx];
  const hipFloatComplex t_y = y[idx];

  f_x[threadIdx.x * 2 + 0] = hipCrealf(t_x);
  f_x[threadIdx.x * 2 + 1] = hipCimagf(t_x);
  f_y[threadIdx.x * 2 + 0] = hipCrealf(t_y);
  f_y[threadIdx.x * 2 + 1] = hipCimagf(t_y);

  // Cast to appropriate types
  // cuFloatComplex *s_x = (cuFloatComplex *)f_x;
  // cuFloatComplex *s_y = (cuFloatComplex *)f_y;
  //
  // s_x[threadIdx.x] = x[idx];
  // s_y[threadIdx.x] = y[idx];
  __syncthreads();

  // add and then write out
  z[idx] = hipCaddf(
      make_hipFloatComplex(f_x[threadIdx.x * 2 + 0], f_x[threadIdx.x * 2 + 1]),
      make_hipFloatComplex(f_y[threadIdx.x * 2 + 0], f_y[threadIdx.x * 2 + 1]));
}

/*
In this one we split the data into separate real/complex arrays.

Consider the stores from global mem -> shared mem.
Under the ordinary packed format, the warp would request for 32*8=256 bytes,
which would be serviced in 2 memory transactions.

This would essentially store indices 0-15, then 16-31, in 2 consecutive rows.

During the addition step, two separate things must happen.
1) The request from the warp is again split into 2 transactions.
2) The addition instruction is then executed for reals,
3) and then for imaginary numbers.

What is unclear is whether the addition instructions are executed after each
transaction, or they are amalgamated after both transactions.

In the former, 4 addition instructions would be needed (1 for each memory
transaction, and for real/imag).

In the latter, only 2 addition instruction would be needed?

Without some deeper investigation into the assembly, it may be difficult to
discern what would happen.



Instead, this kernel splits the reals and imaginaries into two separate arrays
after reading from global mem in a contiguous manner.

There would still be 2 memory transactions, 1 for the reals and 1 for the imags.

During the addition, it would be obvious that the warp would be able to
access all 32 reals without bank conflicts, and similarly for all 32 imags.

Then 2 addition instructions are executed, 1 for the reals and 1 for the imags.

So in this case it feels more explicit(?) that the instruction count is lower.
*/
__global__ void cplxAddDeinterleavedShared(const hipFloatComplex *x,
                                           const hipFloatComplex *y,
                                           hipFloatComplex *z, int n) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // assign shared memory things
  extern __shared__ double s[];

  // allocate each workspace to be same as blockDim
  // leave a space of 32-bits after the first one
  float *s_x_r = (float *)s; // we use float here to count elements
  float *s_x_i = (float *)&s_x_r[blockDim.x];
  float *s_y_r = (float *)&s_x_i[blockDim.x];
  float *s_y_i = (float *)&s_y_r[blockDim.x];

  // Read to thread-local contiguously first
  hipFloatComplex s_x = x[idx];

  // Then split
  s_x_r[threadIdx.x] = s_x.x;
  s_x_i[threadIdx.x] = s_x.y;

  // Similar for y
  hipFloatComplex s_y = y[idx];

  s_y_r[threadIdx.x] = s_y.x;
  s_y_i[threadIdx.x] = s_y.y;

  __syncthreads();

  // add locally first and then write out
  float z_r = s_x_r[threadIdx.x] + s_y_r[threadIdx.x];
  float z_i = s_x_i[threadIdx.x] + s_y_i[threadIdx.x];

  z[idx] = make_hipFloatComplex(z_r, z_i);
}
int main() {
  constexpr int LENGTH = 1000000;

  // Raw allocation
  hipFloatComplex *d_x, *d_y, *d_z, *h_x, *h_y, *h_z;
  hipMalloc((void **)&d_x, sizeof(hipFloatComplex) * LENGTH);
  hipMalloc((void **)&d_y, sizeof(hipFloatComplex) * LENGTH);
  hipMalloc((void **)&d_z, sizeof(hipFloatComplex) * LENGTH);
  hipHostMalloc((void **)&h_x, sizeof(hipFloatComplex) * LENGTH, hipHostMallocDefault);
  hipHostMalloc((void **)&h_y, sizeof(hipFloatComplex) * LENGTH, hipHostMallocDefault);
  hipHostMalloc((void **)&h_z, sizeof(hipFloatComplex) * LENGTH, hipHostMallocDefault);

  // Fill some values
  for (int i = 0; i < LENGTH; ++i) {
    h_x[i] = make_hipFloatComplex((float)i, (float)i);
    h_y[i] = make_hipFloatComplex((float)i, (float)i);
  }

  // Raw copy
  hipMemcpy(d_x, h_x, sizeof(hipFloatComplex) * LENGTH, hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, sizeof(hipFloatComplex) * LENGTH, hipMemcpyHostToDevice);

  // kernels
  constexpr int THREADS_PER_BLK = 128;
  const int numBlks = LENGTH / THREADS_PER_BLK;
  printf("numBlks = %d\n", numBlks);
  // in all cases overallocate the shared mem

  // first kernel
  cplxAddPackedShared<<<numBlks, THREADS_PER_BLK,
                        THREADS_PER_BLK * sizeof(hipFloatComplex) * 3>>>(
      d_x, d_y, d_z, LENGTH);

  // Raw copy back
  hipMemcpy(h_z, d_z, sizeof(hipFloatComplex) * LENGTH, hipMemcpyDeviceToHost);

  for (int i = 0; i < 5; ++i) {
    printf("%f, %f + %f, %f = %f, %f\n", h_x[i].x, h_x[i].y, h_y[i].x, h_y[i].y,
           h_z[i].x, h_z[i].y);

    // reset values
    h_z[i].x = 0.0f;
    h_z[i].y = 0.0f;
  }

  // second kernel
  cplxAddStaggeredShared<<<numBlks, THREADS_PER_BLK,
                           THREADS_PER_BLK * sizeof(hipFloatComplex) * 3>>>(
      d_x, d_y, d_z, LENGTH);
  // Raw copy back
  hipMemcpy(h_z, d_z, sizeof(hipFloatComplex) * LENGTH, hipMemcpyDeviceToHost);

  for (int i = 0; i < 5; ++i) {
    printf("%f, %f + %f, %f = %f, %f\n", h_x[i].x, h_x[i].y, h_y[i].x, h_y[i].y,
           h_z[i].x, h_z[i].y);

    // reset values
    h_z[i].x = 0.0f;
    h_z[i].y = 0.0f;
  }

  // third kernel
  cplxAddDeinterleavedShared<<<numBlks, THREADS_PER_BLK,
                               THREADS_PER_BLK * sizeof(hipFloatComplex) * 3>>>(
      d_x, d_y, d_z, LENGTH);
  // Raw copy back
  hipMemcpy(h_z, d_z, sizeof(hipFloatComplex) * LENGTH, hipMemcpyDeviceToHost);

  for (int i = 0; i < 5; ++i) {
    printf("%f, %f + %f, %f = %f, %f\n", h_x[i].x, h_x[i].y, h_y[i].x, h_y[i].y,
           h_z[i].x, h_z[i].y);

    // reset values
    h_z[i].x = 0.0f;
    h_z[i].y = 0.0f;
  }
  // cleanup
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);

  std::cout << "Complete\n";

  return 0;
}
