#include "hip/hip_runtime.h"
#include "hip/hip_complex.h"
#include "hip/hip_runtime.h"

/*
This is the simple method of storing complex arrays into shared mem.
It simply packs everything in as-is.
*/
__global__ void cplxAddPackedShared(const hipComplex *x, const hipComplex *y,
                                    hipComplex *z, int n) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // assign shared memory things
  extern __shared__ double s[];

  // allocate each workspace to be same as blockDim
  // align one after another, i.e. well packed
  hipComplex *s_x = (hipComplex *)s;
  hipComplex *s_y = (hipComplex *)&s_x[blockDim.x];

  s_x[threadIdx.x] = x[idx];
  s_y[threadIdx.x] = y[idx];
  __syncthreads();

  // add and then write out
  z[idx] = s_x[idx] + s_y[idx];
}

/*
clang-format off

In this one we add a stagger between the two arrays in shared memory.
Assuming a blockDim of multiples of 32, we would have

  0| 1| 2|...................|30|31
 re|im|re|im.................|re|im
.......................
.......................
NUL|re|im| ................        -> this line is the start of y, which is offset by 1 bank

In hindsight, I think this doesn't matter because during the writes are
separate, and the reads are also separate

clang-format on
*/
__global__ void cplxAddStaggeredShared(const hipComplex *x, const hipComplex *y,
                                       hipComplex *z, int n) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // assign shared memory things
  extern __shared__ double s[];

  // allocate each workspace to be same as blockDim
  // leave a space of 32-bits after the first one
  float *f_x = (float *)s; // we use float here to count elements
  float *f_gap = (float *)&f_x[2 * blockDim.x];
  float *f_y = (float *)&f_gap[1];

  // Cast to appropriate types
  hipComplex *s_x = (hipComplex *)f_x;
  hipComplex *s_y = (hipComplex *)f_y;

  s_x[threadIdx.x] = x[idx];
  s_y[threadIdx.x] = y[idx];
  __syncthreads();

  // add and then write out
  z[idx] = s_x[idx] + s_y[idx];
}

/*
In this one we split the data into separate real/complex arrays.

Consider the stores from global mem -> shared mem.
Under the ordinary packed format, the warp would request for 32*8=256 bytes,
which would be serviced in 2 memory transactions.

This would essentially store indices 0-15, then 16-31, in 2 consecutive rows.

During the addition step, two separate things must happen.
1) The request from the warp is again split into 2 transactions.
2) The addition instruction is then executed for reals,
3) and then for imaginary numbers.

What is unclear is whether the addition instructions are executed after each
transaction, or they are amalgamated after both transactions.

In the former, 4 addition instructions would be needed (1 for each memory
transaction, and for real/imag).

In the latter, only 2 addition instruction would be needed?

Without some deeper investigation into the assembly, it may be difficult to
discern what would happen.



Instead, this kernel splits the reals and imaginaries into two separate arrays
after reading from global mem in a contiguous manner.

There would still be 2 memory transactions, 1 for the reals and 1 for the imags.

During the addition, it would be obvious that the warp would be able to
access all 32 reals without bank conflicts, and similarly for all 32 imags.

Then 2 addition instructions are executed, 1 for the reals and 1 for the imags.

So in this case it feels more explicit(?) that the instruction count is lower.
*/
__global__ void cplxAddDeinterleavedShared(const hipComplex *x,
                                           const hipComplex *y, hipComplex *z,
                                           int n) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // assign shared memory things
  extern __shared__ double s[];

  // allocate each workspace to be same as blockDim
  // leave a space of 32-bits after the first one
  float *s_x_r = (float *)s; // we use float here to count elements
  float *s_x_i = (float *)&s_x_r[blockDim.x];
  float *s_y_r = (float *)&s_x_i[blockDim.x];
  float *s_y_i = (float *)&s_y_r[blockDim.x];

  // Read to thread-local contiguously first
  float s_x = x[idx];

  // Then split
  s_x_r[threadIdx.x] = s_x.real();
  s_x_i[threadIdx.x] = s_x.imag();

  // Similar for y
  float s_y = y[idx];

  s_y_r[threadIdx.x] = s_y.real();
  s_y_i[threadIdx.x] = s_y.imag();

  __syncthreads();

  // add locally first and then write out
  float z_r = s_x_r[threadIdx.x] + s_y_r[threadIdx.x];
  float z_i = s_x_i[threadIdx.x] + s_y_i[threadIdx.x];

  z[idx] = hipComplex(z_r, z_i);
}
int main() {
  constexpr int LENGTH = 1000000;

  // Raw allocation
  hipComplex *d_x, *d_y, *d_z, *h_x, *h_y, *h_z;
  hipMalloc((void **)&d_x, sizeof(hipComplex) * LENGTH);
  hipMalloc((void **)&d_y, sizeof(hipComplex) * LENGTH);
  hipMalloc((void **)&d_z, sizeof(hipComplex) * LENGTH);
  hipHostMalloc((void **)&h_x, sizeof(hipComplex) * LENGTH);
  hipHostMalloc((void **)&h_y, sizeof(hipComplex) * LENGTH);
  hipHostMalloc((void **)&h_z, sizeof(hipComplex) * LENGTH);

  // Raw copy
  hipMemcpy(d_x, h_x, sizeof(hipComplex) * LENGTH, hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, sizeof(hipComplex) * LENGTH, hipMemcpyHostToDevice);

  // kernels
  constexpr int THREADS_PER_BLK = 128;
  const int numBlks = LENGTH / THREADS_PER_BLK;
  cplxAddPackedShared<<<numBlks, THREADS_PER_BLK, >>>(d_x, d_y, d_z, LENGTH);
  cplxAddStaggeredShared<<<numBlks, THREADS_PER_BLK, >>>(d_x, d_y, d_z, LENGTH);

  // Raw copy back
  hipMemcpy(h_z, d_z, sizeof(hipComplex) * LENGTH, hipMemcpyDeviceToHost);

  // cleanup
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);

  return 0;
}
