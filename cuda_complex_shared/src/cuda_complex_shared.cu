#include "hip/hip_runtime.h"
#include "hip/hip_complex.h"
#include "hip/hip_runtime.h"

__global__ void cplxAddPackedShared(const hipComplex *x, const hipComplex *y,
                                    hipComplex *z, int n) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // assign shared memory things
  extern __shared__ double s[];

  // allocate each workspace to be same as blockDim
  // align one after another, i.e. well packed
  hipComplex *s_x = (hipComplex *)s;
  hipComplex *s_y = (hipComplex *)&s_x[blockDim.x];

  s_x[threadIdx.x] = x[idx];
  s_y[threadIdx.x] = y[idx];
  __syncthreads();

  // add and then write out
  z[idx] = s_x[idx] * s_y[idx];
}

__global__ void cplxAddStaggeredShared(const hipComplex *x, const hipComplex *y,
                                       hipComplex *z, int n) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // assign shared memory things
  extern __shared__ double s[];

  // allocate each workspace to be same as blockDim
  // leave a space of 32-bits after the first one
  float *f_x = (float *)s; // we use float here to count elements
  float *f_gap = (float *)&f_x[2 * blockDim.x];
  float *f_y = (float *)&f_gap[1];

  // Cast to appropriate types
  hipComplex *s_x = (hipComplex *)f_x;
  hipComplex *s_y = (hipComplex *)f_y;

  s_x[threadIdx.x] = x[idx];
  s_y[threadIdx.x] = y[idx];
  __syncthreads();

  // add and then write out
  z[idx] = s_x[idx] * s_y[idx];
}

int main() {
  constexpr int LENGTH = 1000000;

  // Raw allocation
  hipComplex *d_x, *d_y, *d_z, *h_x, *h_y, *h_z;
  hipMalloc((void **)&d_x, sizeof(hipComplex) * LENGTH);
  hipMalloc((void **)&d_y, sizeof(hipComplex) * LENGTH);
  hipMalloc((void **)&d_z, sizeof(hipComplex) * LENGTH);
  hipHostMalloc((void **)&h_x, sizeof(hipComplex) * LENGTH);
  hipHostMalloc((void **)&h_y, sizeof(hipComplex) * LENGTH);
  hipHostMalloc((void **)&h_z, sizeof(hipComplex) * LENGTH);

  // Raw copy
  hipMemcpy(d_x, h_x, sizeof(hipComplex) * LENGTH, hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, sizeof(hipComplex) * LENGTH, hipMemcpyHostToDevice);

  // kernels
  constexpr int THREADS_PER_BLK = 128;
  const int numBlks = LENGTH / THREADS_PER_BLK;
  cplxAddPackedShared<<<numBlks, THREADS_PER_BLK, >>>(d_x, d_y, d_z, LENGTH);
  cplxAddStaggeredShared<<<numBlks, THREADS_PER_BLK, >>>(d_x, d_y, d_z, LENGTH);

  // Raw copy back
  hipMemcpy(h_z, d_z, sizeof(hipComplex) * LENGTH, hipMemcpyDeviceToHost);

  // cleanup
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);

  return 0;
}
